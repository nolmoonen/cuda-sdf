#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>
#include <sutil/vec_math.h>
#include <sutil/random.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <stb/stb_image_write.h>

#include "vec_math_helper.h"
#include "camera.h"
#include "trace.h"
#include "cuda_check.h"

/// Divide N by S, round up result.
#define ROUND_UP(N, S) ((((N) + (S) - 1) / (S)))

/// SDF of a box.
__forceinline__ __device__ float sd_box(float3 p, float3 b)
{
    float3 q = fabsf(p) - b;
    return length(fmaxf(q, make_float3(0.f))) +
           fminf(fmaxf(q.x, fmaxf(q.y, q.z)), 0.f);
}

/// Returns the distance to a unit-sized menger sponge.
/// https://iquilezles.org/www/articles/menger/menger.htm
__forceinline__ __device__ float map(float3 p)
{
    // start with a unit box
    float d = sd_box(p, make_float3(1.f));

    float scale = 1.f;
    for (int m = 0; m < ITER_COUNT; m++) {
        // optimized, negative part of scaled sdf of three infinite boxes
        float3 a = mod(p * scale, 2.f) - make_float3(1.f);
        scale *= 3.f;
        float3 r = fabsf(make_float3(1.f) - 3.f * fabsf(a));

        float da = fmaxf(r.x, r.y);
        float db = fmaxf(r.y, r.z);
        float dc = fmaxf(r.z, r.x);
        float c = (fminf(da, fminf(db, dc)) - 1.f) / scale;

        d = fmaxf(d, c);
    }

    return d;
}

struct hit {
    float3 hitpoint;
    /// Whether the SDF is hit.
    int hit;
    float3 normal;
};

/// March a ray and return the hitpoint with the SDF.
__forceinline__ __device__ hit trace(float3 origin, float3 direction)
{
    // slight offset to prevent self-intersection
#define TMIN .01f
#define TMAX 1000.f
    hit h;
    for (float t = TMIN; t < TMAX;) {
        float3 p = origin + t * direction;
        float d = map(p);
        if (d < .001f) {
            h.hit = true;
            h.hitpoint = p;
            // find normal using central differences
            // https://iquilezles.org/www/articles/normalsSDF/normalsSDF.htm
            const float eps = .0001f;
            h.normal = normalize(make_float3(
                    map(p + make_float3(eps, 0.f, 0.f)) -
                    map(p - make_float3(eps, 0.f, 0.f)),
                    map(p + make_float3(0.f, eps, 0.f)) -
                    map(p - make_float3(0.f, eps, 0.f)),
                    map(p + make_float3(0.f, 0.f, eps)) -
                    map(p - make_float3(0.f, 0.f, eps))));
            return h;
        }
        // advance the ray with the distance to the sdf, since we know that we
        // won't skip intersections doing this
        t += d;
    }

    h.hit = false;
    return h;
#undef TMAX
#undef TMIN
}

/// Generates a radiance value for the ith sample of this pixel.
__forceinline__ __device__ float3 generate_pixel(
        uint image_idx, uint image_idx_x, uint image_idx_y, uint sample_idx,
        uint size_x, uint size_y, camera *camera)
{
    // initialize random based on sample index and image index
    uint seed = tea<16>(image_idx, sample_idx);

    // generate a ray though the pixel, randomly offset within the pixel
    float2 jitter = make_float2(rnd(seed), rnd(seed));
    float2 res = make_float2(size_x, size_y);
    float2 idx = make_float2(image_idx_x, image_idx_y);
    float2 d = ((idx + jitter) / res) * 2.f - 1.f; // position on raster
    float3 ray_origin = camera->origin;
    float3 ray_direction = normalize(
            d.x * camera->u + d.y * camera->v + camera->w);

    float3 throughput = make_float3(1.f);
    float3 radiance = make_float3(0.f);

    // keep bounding until the maximum number of bounces is hit,
    // or the ray does not intersect with the sdf
    for (int i = 0; i < MAX_BOUNCE_COUNT; i++) {
        hit h = trace(ray_origin, ray_direction);

        if (!h.hit) {
            // 'sky' color
            const float3 color = make_float3(.6, .8f, 1.f);
            radiance += throughput * color;
            break;
        }

        // pick a static diffuse color
        float3 diff_color = make_float3(.2f, .3f, .9f);

        // check if we continue using russian roulette, where the max component
        // of the color dictates the probability
        if (i > MIN_BOUNCE_COUNT) {
            float rr_prob = fmaxf(diff_color);
            if (rr_prob < rnd(seed)) break;
            // if continuing, scale with probability
            throughput /= rr_prob;
        }

        // surface model is lambertian, attenuation is equal to diffuse
        // color, assuming we sampled with cosine weighted hemisphere
        throughput *= diff_color;

        // set new origin and generate new direction
        ray_origin = h.hitpoint;
        float3 w_in = cosine_sample_hemisphere(rnd(seed), rnd(seed));
        frame onb(h.normal);
        onb.inverse_transform(w_in);
        ray_direction = w_in;
    }

    return radiance;
}

/// Implementation with regeneration: create a number of persistent threads that
/// complete samples one by one, starting new ones when the current one is
/// terminated.
__global__ void generate_pixel_regeneration(
        uint size_x, uint size_y, uint sample_count, float *buffer,
        camera *camera, unsigned long long int *idx)
{
    const ulong max_count = size_x * size_y * sample_count;
    while (true) {
        // obtain the next index. if is it out of bounds, stop
        unsigned long long int this_idx = atomicAdd(idx, 1);
        if (this_idx >= max_count) break;

        uint sample_idx = this_idx / (size_x * size_y);
        uint image_idx = this_idx - sample_idx * size_x * size_y;
        uint image_idx_y = image_idx / size_x;
        uint image_idx_x = image_idx - image_idx_y * size_x;

        // obtain radiance
        float3 radiance = generate_pixel(
                image_idx, image_idx_x, image_idx_y, sample_idx,
                size_x, size_y, camera);

        // atomically add to buffer
        atomicAdd(&buffer[4 * image_idx + 0], radiance.x / float(sample_count));
        atomicAdd(&buffer[4 * image_idx + 1], radiance.y / float(sample_count));
        atomicAdd(&buffer[4 * image_idx + 2], radiance.z / float(sample_count));
    }
}

/// Naive implementation: create a number of threads at least equal to the
/// number of pixels * the number of samples, each thread computes all samples
/// for that pixel.
__global__ void generate_pixel_naive(
        uint size_x, uint size_y, uint sample_count, float *buffer,
        camera *camera)
{
    uint2 idx = make_uint2(
            blockDim.x * blockIdx.x + threadIdx.x,
            blockDim.y * blockIdx.y + threadIdx.y);

    if (idx.x >= size_x || idx.y >= size_y) return;
    const uint image_idx = idx.y * size_x + idx.x;

    float3 accumulated_color = make_float3(0.f);

    for (int i = 0; i < sample_count; i++) {
        float3 radiance = generate_pixel(
                image_idx, idx.x, idx.y, i, size_x, size_y, camera);

        accumulated_color += radiance / float(sample_count);
    }

    // write to buffer
    buffer[4 * image_idx + 0] = accumulated_color.x;
    buffer[4 * image_idx + 1] = accumulated_color.y;
    buffer[4 * image_idx + 2] = accumulated_color.z;
}

/// Converts a linear radiance value to a sRGB pixel value.
uchar radiance_to_srgb(float val)
{ return (uchar) (clamp(powf(val, 1.f / 2.4f), 0.f, 1.f) * 255.f); }

void generate(
        uint size_x, uint size_y, uint sample_count, const char *filename)
{
    // variables that define the camera
    float3 origin = make_float3(-.3f);
    float3 target = make_float3(1.f);
    float3 up = make_float3(0.f, 1.f, 0.f);

    // initialize camera based on the above variables
    float aspect = float(size_x) / float(size_y);
    camera cam;
    cam.origin = origin;
    cam.w = normalize(target - cam.origin);       // lookat direction
    cam.u = normalize(cross(cam.w, up)) * aspect; // screen right
    cam.v = normalize(cross(cam.u, cam.w));       // screen up

    // copy camera parameters to device
    camera *d_cam = nullptr;
    CUDA_CHECK(hipMalloc(&d_cam, sizeof(camera)));
    CUDA_CHECK(hipMemcpy(d_cam, &cam, sizeof(camera), hipMemcpyHostToDevice));

    // create output buffer on device
    float *d_buffer = nullptr;
    size_t buffer_size = sizeof(float) * 4 * size_x * size_y;
    CUDA_CHECK(hipMalloc(&d_buffer, buffer_size));
    CUDA_CHECK(hipMemset(d_buffer, 0, buffer_size));

    // events for measuring elapsed time
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // declare bool to get warnings even in unused code
#ifdef NAIVE
    bool do_naive = true;
#else
    bool do_naive = false;
#endif

    // launch kernel
    CUDA_CHECK(hipEventRecord(start));
    unsigned long long int *d_idx = nullptr;
    if (do_naive) {
#define BLOCK_SIZE 16
        dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 block_count(
                ROUND_UP(size_x, BLOCK_SIZE), ROUND_UP(size_y, BLOCK_SIZE), 1);
        generate_pixel_naive<<<block_count, block_size>>>(
                size_x, size_y, sample_count, d_buffer, d_cam);
#undef BLOCK_SIZE
    } else {
        // additionally, allocate a single long int counter
        CUDA_CHECK(hipMalloc(&d_idx, sizeof(unsigned long long int)));
        CUDA_CHECK(hipMemset(d_idx, 0, sizeof(unsigned long long int)));
        generate_pixel_regeneration<<<1024, 1024>>>(
                size_x, size_y, sample_count, d_buffer, d_cam, d_idx);
    }
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0.f;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("kernel took %fs\n", milliseconds * 1e-3f);

    // when kernel is done, copy buffer back to host
    CUDA_SYNC_CHECK();
    if (!do_naive) CUDA_CHECK(hipFree(d_idx));
    CUDA_CHECK(hipFree(d_cam));
    float *buffer = (float *) malloc(buffer_size);
    CUDA_CHECK(hipMemcpy(
            buffer, d_buffer, buffer_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_buffer));

    // convert buffer to format accepted by image writer
    uchar *image = (uchar *) malloc(sizeof(char) * 3 * size_x * size_y);
    for (uint i = 0; i < size_x * size_y; i++) {
        image[3 * i + 0] = radiance_to_srgb(buffer[4 * i + 0]);
        image[3 * i + 1] = radiance_to_srgb(buffer[4 * i + 1]);
        image[3 * i + 2] = radiance_to_srgb(buffer[4 * i + 2]);
    }
    free(buffer);

    // write buffer to file
    stbi_flip_vertically_on_write(1);
    stbi_write_png(filename, size_x, size_y, 3, image, size_x * 3);
    free(image);
}
