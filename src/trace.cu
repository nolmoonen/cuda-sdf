#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>
#include <deps/sutil/vec_math.h>
#include <deps/sutil/random.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <deps/stb/stb_image_write.h>

#include "vec_math_helper.h"
#include "camera.h"
#include "trace.h"
#include "cuda_check.h"

/// Divide N by S, round up result.
#define ROUND_UP(N, S) ((((N) + (S) - 1) / (S)))

__forceinline__ __device__ float sd_box(float3 p, float3 b)
{
    float3 q = fabsf(p) - b;
    return length(fmaxf(q, make_float3(0.f))) +
           fminf(fmaxf(q.x, fmaxf(q.y, q.z)), 0.f);
}

/// Returns the distance to a unit-sized menger sponge and a color value based
/// on the number of iterations of the closest surface.
/// https://iquilezles.org/www/articles/menger/menger.htm
__forceinline__ __device__ float2 map(float3 p)
{
    float d = sd_box(p, make_float3(1.f));
    float col = 1.f;

    float s = 1.f;
    for (int m = 0; m < ITER_COUNT; m++) {
        float3 a = mod(p * s, 2.f) - make_float3(1.f);
        s *= 3.f;
        float3 r = fabsf(make_float3(1.f) - 3.f * fabsf(a));

        float da = fmaxf(r.x, r.y);
        float db = fmaxf(r.y, r.z);
        float dc = fmaxf(r.z, r.x);
        float c = (fminf(da, fminf(db, dc)) - 1.f) / s;

        if (c > d) {
            d = c;
            // assign a color based on the iteration count
            col = (1.f + float(m)) / float(ITER_COUNT + 1);
        }
    }

    return make_float2(d, col);
}

struct hit {
    float3 hitpoint;
    /// Whether the SDF is hit.
    int hit;
    float3 normal;
    /// Single-valued color.
    float color;
};

__forceinline__ __device__ hit trace(float3 origin, float3 direction)
{
    // slight offset to prevent self-intersection
#define TMIN .01f
#define TMAX 1000.f
    hit h;
    for (float t = TMIN; t < TMAX;) {
        float3 p = origin + t * direction;
        float2 d = map(p);
        if (d.x < .001f) {
            h.hit = true;
            h.hitpoint = p;
            // find normal using central differences
            // https://iquilezles.org/www/articles/normalsSDF/normalsSDF.htm
            const float eps = .0001f;
            h.normal = normalize(make_float3(
                    map(p + make_float3(eps, 0.f, 0.f)).x -
                    map(p - make_float3(eps, 0.f, 0.f)).x,
                    map(p + make_float3(0.f, eps, 0.f)).x -
                    map(p - make_float3(0.f, eps, 0.f)).x,
                    map(p + make_float3(0.f, 0.f, eps)).x -
                    map(p - make_float3(0.f, 0.f, eps)).x));
            h.color = d.y;
            return h;
        }
        // advance the ray with the distance to the sdf, since we know that we
        // won't skip intersections doing this
        t += d.x;
    }

    h.hit = false;
    return h;
#undef TMAX
#undef TMIN
}

/// Generates a radiance value for the ith sample of this pixel.
__forceinline__ __device__ float3 generate_pixel(
        uint image_idx, uint image_idx_x, uint image_idx_y, uint sample_idx,
        uint size_x, uint size_y, camera *camera)
{
    // initialize random based on sample index and image index
    uint seed = tea<16>(image_idx, sample_idx);

    // generate a ray though the pixel, randomly offset within the pixel
    float2 jitter = make_float2(rnd(seed), rnd(seed));
    float2 res = make_float2(size_x, size_y);
    float2 idx = make_float2(image_idx_x, image_idx_y);
    float2 d = ((idx + jitter) / res) * 2.f - 1.f; // position on raster
    float3 ray_origin = camera->origin;
    float3 ray_direction = normalize(
            d.x * camera->u + d.y * camera->v + camera->w);

    float3 throughput = make_float3(1.f);
    float3 radiance = make_float3(0.f);

    // keep bounding until the maximum number of bounces is hit,
    // or the ray does not intersect with the sdf
    for (int i = 0; i < BOUNCE_COUNT; i++) {
        hit h = trace(ray_origin, ray_direction);

        if (!h.hit) {
            // 'sky' color
            const float3 color = make_float3(.6, .8f, 1.f);
            radiance += throughput * color;
            break;
        }

        // find a diffuse color based on the single color value
        const float3 diff_color = make_float3(
                (1.f - h.color) * .5f, (1.f - h.color) * .3f,
                h.color * .6f);

        // surface model is lambertian, attenuation is equal to diffuse
        // color, assuming we sampled with cosine weighted hemisphere
        throughput *= diff_color;

        // set new origin and generate new direction
        ray_origin = h.hitpoint;
        float3 w_in = cosine_sample_hemisphere(rnd(seed), rnd(seed));
        frame onb(h.normal);
        onb.inverse_transform(w_in);
        ray_direction = w_in;
    }

    return radiance;
}

/// Implementation with regeneration: create a number of persistent threads that
/// complete samples one by one, starting new ones when the current one is
/// terminated.
__global__ void generate_pixel_regeneration(
        uint size_x, uint size_y, uint sample_count, float *buffer,
        camera *camera, ulong *idx)
{
    const ulong max_count = size_x * size_y * sample_count;
    while (true) {
        // obtain the next index. if is it out of bounds, stop
        ulong this_idx = atomicAdd(idx, 1);
        if (this_idx >= max_count) break;

        uint sample_idx = this_idx / (size_x * size_y);
        uint image_idx = this_idx - sample_idx * size_x * size_y;
        uint image_idx_y = image_idx / size_x;
        uint image_idx_x = image_idx - image_idx_y * size_x;

        // obtain radiance
        float3 radiance = generate_pixel(
                image_idx, image_idx_x, image_idx_y, sample_idx,
                size_x, size_y, camera);

        // atomically add to buffer
        atomicAdd(&buffer[4 * image_idx + 0], radiance.x / float(sample_count));
        atomicAdd(&buffer[4 * image_idx + 1], radiance.y / float(sample_count));
        atomicAdd(&buffer[4 * image_idx + 2], radiance.z / float(sample_count));
    }
}

/// Naive implementation: create a number of threads at least equal to the
/// number of pixels * the number of samples, each thread computes all samples
/// for that pixel.
__global__ void generate_pixel_naive(
        uint size_x, uint size_y, uint sample_count, float *buffer,
        camera *camera)
{
    uint2 idx = make_uint2(
            blockDim.x * blockIdx.x + threadIdx.x,
            blockDim.y * blockIdx.y + threadIdx.y);

    if (idx.x >= size_x || idx.y >= size_y) return;
    const uint image_idx = idx.y * size_x + idx.x;

    float3 accumulated_color = make_float3(0.f);

    for (int i = 0; i < sample_count; i++) {
        float3 radiance = generate_pixel(
                image_idx, idx.x, idx.y, i, size_x, size_y, camera);

        accumulated_color += radiance / float(sample_count);
    }

    // write to buffer
    buffer[4 * image_idx + 0] = accumulated_color.x;
    buffer[4 * image_idx + 1] = accumulated_color.y;
    buffer[4 * image_idx + 2] = accumulated_color.z;
}

/// Converts a linear radiance value to a sRGB pixel value.
uchar radiance_to_srgb(float val)
{ return (uchar) (clamp(powf(val, 1.f / 2.4f), 0.f, 1.f) * 255.f); }

void generate(
        uint size_x, uint size_y, uint sample_count, const char *filename)
{
    // initialize camera
    camera cam;
    cam.origin = make_float3(2.1f, 0.f, 0.f);
    float3 target = make_float3(0.f);
    const float3 up = make_float3(0.f, 1.f, 0.f);
    float aspect = float(size_x) / float(size_y);
    cam.w = normalize(target - cam.origin); // lookat direction
    cam.u = normalize(cross(cam.w, up)) * aspect; // screen right
    cam.v = normalize(cross(cam.u, cam.w)); // screen up

    // copy camera parameters to device
    camera *d_cam = nullptr;
    CUDA_CHECK(hipMalloc(&d_cam, sizeof(camera)));
    CUDA_CHECK(hipMemcpy(d_cam, &cam, sizeof(camera), hipMemcpyHostToDevice));

    // create output buffer on device
    float *d_buffer = nullptr;
    size_t buffer_size = sizeof(float) * 4 * size_x * size_y;
    CUDA_CHECK(hipMalloc(&d_buffer, buffer_size));
    CUDA_CHECK(hipMemset(d_buffer, 0, buffer_size));

    // events for measuring elapsed time
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // declare bool to get warnings even in unused code
#ifdef NAIVE
    bool do_naive = true;
#else
    bool do_naive = false;
#endif

    // launch kernel
    CUDA_CHECK(hipEventRecord(start));
    ulong *d_idx = nullptr;
    if (do_naive) {
#define BLOCK_SIZE 16
        dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 block_count(
                ROUND_UP(size_x, BLOCK_SIZE), ROUND_UP(size_y, BLOCK_SIZE), 1);
        generate_pixel_naive<<<block_count, block_size>>>(
                size_x, size_y, sample_count, d_buffer, d_cam);
#undef BLOCK_SIZE
    } else {
        // additionally, allocate a single long int counter
        CUDA_CHECK(hipMalloc(&d_idx, sizeof(ulong)));
        CUDA_CHECK(hipMemset(d_idx, 0, sizeof(ulong)));
        generate_pixel_regeneration<<<128, 512>>>(
                size_x, size_y, sample_count, d_buffer, d_cam, d_idx);
    }
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0.f;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("kernel took %fs\n", milliseconds * 1e-3f);

    // when kernel is done, copy buffer back to host
    CUDA_SYNC_CHECK();
    if (!do_naive) CUDA_CHECK(hipFree(d_idx));
    CUDA_CHECK(hipFree(d_cam));
    float *buffer = (float *) malloc(buffer_size);
    CUDA_CHECK(hipMemcpy(
            buffer, d_buffer, buffer_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_buffer));

    // convert buffer to format accepted by image writer
    uchar *image = (uchar *) malloc(sizeof(char) * 3 * size_x * size_y);
    for (uint i = 0; i < size_x * size_y; i++) {
        image[3 * i + 0] = radiance_to_srgb(buffer[4 * i + 0]);
        image[3 * i + 1] = radiance_to_srgb(buffer[4 * i + 1]);
        image[3 * i + 2] = radiance_to_srgb(buffer[4 * i + 2]);
    }
    free(buffer);

    // write buffer to file
    stbi_flip_vertically_on_write(1);
    stbi_write_png(filename, size_x, size_y, 3, image, size_x * 3);
    free(image);
}